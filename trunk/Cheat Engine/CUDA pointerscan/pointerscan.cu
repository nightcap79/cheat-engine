#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <windows.h>
#include <stdio.h>

#include "pscanfileaccess.h"
#include "cudapointervaluelist.cuh"

typedef struct _rcaller //recursion replacement
{
	UINT_PTR startvalue;  //0 when not used
	UINT_PTR stopvalue;  //
	PPointerList plist;
	int plistIndex;  
} rcaller, *prcaller;

typedef struct _continueData
{	
	int level;  //the current level	(-1 if not running)
	prcaller caller; //
	int *offsets; //temp offset list
} ContinueData, *PContinueData;

typedef struct _workcommand  //same as continuedata but no plist data
{
	UINT_PTR startvalue;  
	UINT_PTR stopvalue;  
	int level;  
	int *offsets; 
} WorkCommand, *PWorkCommand;

typedef struct _workCommandList
{
	unsigned int count;
	WorkCommand list[16];
} WorkCommandList, *PWorkCommandList;

__device__ WorkCommandList SavedWorkCommandList;
__device__ WorkCommandList SavedWorkCommandList2;


__global__ void pscan(PContinueData cd, int structsize, int maxlevel, PWorkCommandList PreviousSavedCommandList, PWorkCommandList CurrentSavedCommandList)
/*
The pointerscanner loop
*/
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int level;

  printf("pscan index=%d\n", index);

  //check if the current thread should continue from the last spot, or fetch a new work command
  if (cd[index].level==-1)
  {
    //fetch the state from the previous saved commad list
    unsigned int i=atomicDec(&(PreviousSavedCommandList->count), 0); //it never goes below 0
    
    if (i>0)
    {  
      //something was in it
      int level;
      i--;
      level=PreviousSavedCommandList->list[i].level;
      cd[index].level=level;
      
      cd[index].caller[cd[index].level].startvalue=PreviousSavedCommandList->list[i].startvalue;
      cd[index].caller[cd[index].level].stopvalue=PreviousSavedCommandList->list[i].stopvalue;      
      
      //memcpy(cd[index].offsets, PreviousSavedCommandList->list[i].offsets, sizeof(int)*PreviousSavedCommandList->list[i].level); 
            
      printf("Has data: PreviousSavedCommandList->count=%d (i=%d)\n", PreviousSavedCommandList->count, i);
    };    
      
  }
  
  
  level=cd[index].level;
  
  while (level>=0)
  {  
	//continue from the current state
	
	UINT_PTR startvalue=cd[index].caller[level].startvalue;
	UINT_PTR stopvalue=cd[index].caller[level].stopvalue;
	PPointerList plist=NULL;
	
	printf("%d: Processing\n", index);
	printf("%d: StartValue:%x\n", index, (unsigned int)startvalue);
	printf("%d: StopValue: %x\n", index, (unsigned int)cd[index].caller[level].stopvalue);
	
	plist=findPointerValue(startvalue, &stopvalue);
	printf("plist=%p\n", plist);
	
	printf("stopvalue=%x\n", (unsigned int)stopvalue);
	
	
	//.....
	
	level--;
	if (level>=0)
	{
		if (cd[index].caller[level].startvalue)
		{
		  //it has data, set it up
		  cd[index].level=level;
		}
		else
		{
		  cd[index].level=-1; //end of recursive call reached
		}		
	}
  }





}

__global__ void initpscan(PContinueData cd, UINT_PTR address, int structsize, int maxlevel, UINT_PTR valuetofind)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	cd[index].caller=(prcaller)malloc(sizeof(rcaller)*maxlevel);
	if (index==0)
	{
	    int i;
		printf("initpscan %d\n", index);
	
		cd[index].level=0;
		
		cd[index].caller[0].startvalue=address-structsize;
		cd[index].caller[0].stopvalue=address; 
		
		SavedWorkCommandList.count=0;
		SavedWorkCommandList2.count=0;	
		
		for (i=0; i<16; i++)
		{
			SavedWorkCommandList.list[i].offsets=(int *)malloc(sizeof(int)*maxlevel);
			SavedWorkCommandList2.list[i].offsets=(int *)malloc(sizeof(int)*maxlevel);
		}
	}
	else
	{	
		cd[index].level=-1;		
	}

}


int pointerscan(UINT_PTR address, int structsize, int maxlevel)
{
  PContinueData cd;
  
  PWorkCommandList l1, l2;
  
  
  hipMalloc(&cd, sizeof(ContinueData)*1024);
  initpscan<<<1,1024>>>(cd, address, structsize, maxlevel, address);

  hipDeviceSynchronize(); 
  printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));     


  //loop till all are done
  hipGetSymbolAddress((void **)&l1, SavedWorkCommandList);
  hipGetSymbolAddress((void **)&l2, SavedWorkCommandList2);
  
  pscan<<<1,1>>>(cd, structsize, 5, l1, l2); 
  
  //loop
  
  hipDeviceSynchronize(); 
  printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));    
  
   
    
  
  
  return 0;
}